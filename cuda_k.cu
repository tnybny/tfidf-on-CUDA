#include "hip/hip_runtime.h"
/*BSD License

  Copyright © belongs to the uploader, all rights reserved.

  Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

  Redistributions of source code must retain the above copyright notice, with the name of the uploader, and this list of conditions;

  Redistributions in binary form must reproduce the above copyright notice, with the name of the uploader, and this list of conditions in the documentation and/or other materials provided with the distribution;
  Neither the name of the uploader nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
 */

#include <stdio.h>
#include <math.h>
#include "string_funcs.cu"
//#include "hash_funcs.cpp"
#include "defs.h"
//#include "hash_funcs.h"
//#include "cuda_k.h"
#include "hash_funcs_cuda.h"


char *device_local;
unsigned int *token_length_device;
CalcFreqController *token_division_controller_device;
MyHashMapElement *hash_doc_token_sub_tables_device;
MyHashMapElement *hash_doc_token_tables_device;
MyHashMapElement *occ_hash_table_device;
float *bucket_sqrt_sum_device;
float *doc_similarity_matrix_device;
int *doc_rank_matrix_device;
__device__ bool stripPrefixes ( char *str);
__global__ void StripAffixes(char *dev_res, unsigned int *token_length, CalcFreqController *controller, int docs_count);
__global__ void MakeDocHash2(char *dev_mem, unsigned int *token_length, CalcFreqController *controller,
		MyHashMapElement *hash_doc_token_sub_tables, MyHashMapElement *hash_doc_token_tables, int sub_table_size, int table_size, int maxRows, size_t pitch1, size_t pitch2);
__global__ void MakeDocHash(char *dev_mem, unsigned int *token_length, CalcFreqController *controller,
		MyHashMapElement *hash_doc_token_sub_tables, MyHashMapElement *hash_doc_token_tables, int sub_table_size, int table_size, int maxRows, size_t pitch1, size_t pitch2);
size_t pitch1;
size_t pitch2;
__global__ void AddToOccTable(MyHashMapElement *hash_doc_token_tables, MyHashMapElement *occ_hash_table, int numDocs, size_t pitch2);
float *simbase;
int *rankbase;

#define TRUE 1
#define FALSE 0

#define tablesize PACKET_SIZE

//int strCmp(char *str1, char *str2);

/* This is djb2 hashing algorithm by Dan Bernstien, from comp.lang.c*/
__device__ unsigned long computeHashCuda(char *str)
{
#if 1
	unsigned long hash = 5381;
	int c;

	while (c = *str++)
		hash = ((hash << 5) + hash) + c;  // hash * 33 + c 

	return hash;
#else
	unsigned long hash = 0;
	int c;
	int i = 0;

	while (c = *str++)
	{
		hash = hash * i + c;
		i++;
	}
	return hash;
#endif
}

// my stuff 
__device__ void initHashTableCuda(MyHashMapElement *hme, int tablerange, int subrange)
{
	MyHashMapElement *bucket = hme;
	for (int i = 0; i != tablerange; i++)
	{
		bucket->countInBuc = 0;
		/*dbg{
		  bucket->freq = 0; // TODO not necessary
		  bucket->key = 0xDEADBEAF;
		  bucket->tokenLength = 0;
		  bucket->subkey = 0;
		  for (int j = 0; j < subrange; j++)
		  {
		  (bucket+j)->countInBuc = 0;
		  (bucket+j)->freq = 0;
		  (bucket+j)->key = 0xDEADBEAF;
		  (bucket+j)->tokenLength = 0;
		  }
		  }*/
		bucket += subrange;
	}
}

__device__ bool insertElementCuda(MyHashMapElement *hme, unsigned long key, int keyshift, int bucketsize, int strlength, int initvalue)
{
	unsigned long newkey = key & ( (1 << keyshift) - 1 );  // clear the MSBs
	MyHashMapElement *bucket = &hme[newkey * bucketsize];
	int numEleInBucket = bucket->countInBuc;
	// search if the same element is in the bucket, if in, incr the frequency
	for (int i = 0; i != numEleInBucket; i++)
	{ 
		if (bucket[i].key == key && bucket[i].tokenLength == strlength) 
		{
			bucket[i].freq+=initvalue;
			return true;
		}
	}

	if (numEleInBucket == bucketsize) return false;  // if bucket full, drop the element TODO 

	bucket[0].countInBuc++;
	bucket[numEleInBucket].key = key;
	bucket[numEleInBucket].freq = initvalue;
	bucket[numEleInBucket].tokenLength = strlength;
	dbg{
		bucket[numEleInBucket].subkey = newkey;
		bucket[numEleInBucket].countInBuc = numEleInBucket + 1;
	}
	return true;
	//  bucket[numEleInBucket].docIndex = 
	//  bucket[numEleInBucket].tokenLength = 
}

__device__ int findElementCuda(MyHashMapElement *hme, unsigned long key, int keyshift, int bucketsize, int strlength)
{
	unsigned long newkey = key & ( (1 << keyshift) - 1 );  // clear the MSBs
	MyHashMapElement *bucket = &hme[newkey * bucketsize];
	int numEleInBucket = bucket->countInBuc;
	// search if the same element is in the bucket, if in, incr the frequency
	for (int i = 0; i != numEleInBucket; i++)
	{ 
		if (bucket[i].key == key && bucket[i].tokenLength == strlength) 
			return bucket[i].freq;
	}

	return 0; 
}

__constant__ char prefixes[][16]= { "kilo", "micro", "milli", "intra", "ultra", "mega", "nano", "pico", "pseudo"};

__constant__ char suffixes2[][2][16] = { { "ational", "ate" },
	{ "tional",  "tion" },
	{ "enci",    "ence" },
	{ "anci",    "ance" },
	{ "izer",    "ize" },
	{ "iser",    "ize" },
	{ "abli",    "able" },
	{ "alli",    "al" },
	{ "entli",   "ent" },
	{ "eli",     "e" },
	{ "ousli",   "ous" },
	{ "ization", "ize" },
	{ "isation", "ize" },
	{ "ation",   "ate" },
	{ "ator",    "ate" },
	{ "alism",   "al" },
	{ "iveness", "ive" },
	{ "fulness", "ful" },
	{ "ousness", "ous" },
	{ "aliti",   "al" },
	{ "iviti",   "ive" },
	{ "biliti",  "ble" }};

__constant__ char suffixes3[][2][16] = { { "icate", "ic" },
	{ "ative", "" },
	{ "alize", "al" },
	{ "alise", "al" },
	{ "iciti", "ic" },
	{ "ical",  "ic" },
	{ "ful",   "" },
	{ "ness",  "" }};

__constant__ char suffixes4[][16] = { "al",
	"ance",
	"ence",
	"er",
	"ic",
	"able", "ible", "ant", "ement", "ment", "ent", "sion", "tion",
	"ou", "ism", "ate", "iti", "ous", "ive", "ize", "ise"};


__device__ bool step1(char *str ) {

	char stem[32];
	bool changed = false;
	if ( str[strLen(str)-1] == 's' ) {
		if ( (hasSuffix( str, "sses", stem ))
				|| (hasSuffix( str, "ies", stem)) ){
			str[strLen(str)-2] = '\0';
			changed = true;
		}
		else {
			if ( ( strLen(str) == 1 )
					&& ( str[strLen(str)-1] == 's' ) ) {
				str[0] = '\0';
				return true;
			}
			if ( str[strLen(str)-2 ] != 's' ) {
				str[strLen(str)-1] = '\0';
				changed = true;
			}
		}
	}

	if ( hasSuffix( str,"eed",stem ) ) {
		if ( measure( stem ) > 0 ) {
			str[strLen(str)-1] = '\0';
			changed = true;
		}
	}
	else {
		if (  (hasSuffix( str,"ed",stem ))
				|| (hasSuffix( str,"ing",stem )) ) {
			if (containsVowel( stem ))  {

				if(stem[0]=='\0')
				{
					str[0]='\0';
					changed = true;
				}
				else
				{
					str[strLen(stem)] = '\0';
					changed = true;
				}
				if ( strLen(str) == 1 )
					return changed;

				if ( ( hasSuffix( str,"at",stem) )
						|| ( hasSuffix( str,"bl",stem ) )
						|| ( hasSuffix( str,"iz",stem) ) ) {
					int len = strLen(str);
					str[len-1] = 'e';
					str[len] = '\0';
					changed = true;

				}
				else {
					int length = strLen(str);
					if ( (str[length-1] == str[length-2])
							&& (str[length-1] != 'l')
							&& (str[length-1] != 's')
							&& (str[length-1] != 'z') ) {
						str[length-1]='\0';
						changed = true;
					}
					else
						if ( measure( str ) == 1 ) {
							if ( cvc(str) )
							{
								str[length-1]='e';
								str[length]='\0';
								changed = true;
							}   
						}
				}
			}
		}
	}

	if ( hasSuffix(str,"y",stem) )
		if ( containsVowel( stem ) ) {
			int len = strLen(str);
			str[len-1]='i';
			str[len]='\0';
			changed = true;
		}
	return changed;
}

__device__ bool step2( char *str ) {

	char stem[32];
	int last = sizeof(suffixes2)/(sizeof(char)*2*16); //strange way of calculating length of array
	bool changed = false;

	for ( int index = 0 ; index < last; index++ ) {
		if ( hasSuffix ( str, suffixes2[index][0], stem ) ) {
			if ( measure ( stem ) > 0 ) {
				int stemlen, suffixlen, j;
				stemlen = strLen(stem);
				suffixlen = strLen(suffixes2[index][1]);
				changed = true;
				for(j=0; j<suffixlen; j++)
					str[stemlen+j] = suffixes2[index][1][j];
				str[stemlen+j] = '\0';
			}
		}
	}
	return changed;
}

__device__ bool step3( char *str ) {

	char stem[32];
	int last = sizeof(suffixes3)/(sizeof(char)*2*16); //strange way of calculating length of array/    
	bool changed= false;
	for ( int index = 0 ; index<last; index++ ) {
		if ( hasSuffix ( str, suffixes3[index][0], stem ))
			if ( measure ( stem ) > 0 ) {
				int stemlen, suffixlen, j;
				stemlen = strLen(stem);
				suffixlen = strLen(suffixes3[index][1]);
				changed = true;
				for( j=0; j<suffixlen; j++)
					str[stemlen+j] = suffixes3[index][1][j];
				str[stemlen+j] = '\0';
			}
	}
	return changed ;  
}

__device__ bool step4( char *str ) {

	char stem[32];
	int last = sizeof(suffixes4)/(sizeof(char)*16); //strange way of calculating length of array
	bool changed = false;
	for ( int index = 0 ; index<last; index++ ) {
		if ( hasSuffix ( str, suffixes4[index], stem ) ) {
			changed = true;
			if ( measure ( stem ) > 1 ) {
				str[strLen(stem)] = '\0';
			}
		}
	}
	return changed;
}

__device__ bool step5( char *str ) {

	bool changed = false;
	if ( str[strLen(str)-1] == 'e' ) {
		if ( measure(str) > 1 ) {
			str[strLen(str)-1] = '\0';
			changed = true;
		}
		else
			if ( measure(str) == 1 ) {
				char stem[32];
				int i;
				for ( i=0; i<strLen(str)-1; i++ )
					stem[i] = str[i];
				stem[i] = '\0';

				if ( !cvc(stem) ){
					str[strLen(str)-1] = '\0';
					changed = true;
				}
			}
	}

	if ( strLen(str) == 1 )
		return true;
	if ( (str[strLen(str)-1] == 'l')
			&& (str[strLen(str)-2] == 'l') && (measure(str) > 1) )
		if ( measure(str) > 1 ) {
			str[strLen(str)-1] = '\0';
			changed = true;
		}

	return changed;
}



__device__ bool stripSuffixes(char *str ) {

	bool changed = false;
	changed = step1( str );
	if ( strLen(str) >= 1 )
		changed |= step2( str );
	if ( strLen(str) >= 1 )
		changed |= step3( str );
	if ( strLen(str) >= 1 )
		changed |= step4( str );
	if ( strLen(str) >= 1 )
		changed |= step5( str );
	return changed;
}

__device__ bool stripPrefixes ( char *str) {

	int  newLen, j;
	bool found = false;

	int last = sizeof(prefixes)/(sizeof(char)*16); //strange way of calculating length of array
	for ( int i=0 ; i<last; i++ ) 
	{
		//Find if str starts with prefix prefixes[i]
		found = prefixFind(str, prefixes[i]);
		if (found)
		{
			newLen = strLen(str) - strLen(prefixes[i]);
			for (j=0 ; j < newLen; j++ )
				str[j] = str[j+strLen(prefixes[i])];
			str[j] = '\0';
		}
	}
	return found;
}

void StripAffixesWrapper(char *host_local, unsigned int *token_length_host, CalcFreqController *token_division_controller_host, int tokens_count, int docs_count, int g, int b)
{

	//hipMalloc
	hipMalloc(&device_local, tokens_count * sizeof(char) * TOKEN_MAX_SIZE_PLUS_END);
	hipMalloc(&token_length_device, tokens_count*sizeof(unsigned int));
	hipMalloc(&token_division_controller_device, docs_count * sizeof(CalcFreqController));

	//cuda Mempcpy
	hipMemcpy(device_local, host_local, tokens_count * sizeof(char) * TOKEN_MAX_SIZE_PLUS_END, hipMemcpyHostToDevice);
	hipMemcpy(token_length_device, token_length_host, tokens_count*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(token_division_controller_device, token_division_controller_host, docs_count * sizeof(CalcFreqController), hipMemcpyHostToDevice);

	//kernel call
	StripAffixes<<<g, b>>>(device_local, token_length_device, token_division_controller_device, docs_count);

	//cuda Memcpy
	hipMemcpy(host_local, device_local, tokens_count * sizeof(char) * TOKEN_MAX_SIZE_PLUS_END, hipMemcpyDeviceToHost);
	hipMemcpy(token_length_host, token_length_device, tokens_count*sizeof(unsigned int), hipMemcpyDeviceToHost);

	return;
}

__global__ void StripAffixes(char *dev_res, unsigned int *token_length, CalcFreqController *controller, int docs_count)
{
	int numBags = MAX_THREADS;
	int doc = blockIdx.x;
	int tkn = threadIdx.x;
	if(tkn < MAX_THREADS && doc < docs_count)
	{
		__shared__ char tokens[TOKEN_MAX_SIZE_PLUS_END * MAX_THREADS];
		// adjust the token and token_length array pointer according to controller 
		char *base = &dev_res[controller[doc].doc_token_start * TOKEN_MAX_SIZE_PLUS_END];
		unsigned int *token_length_base = &token_length[controller[doc].doc_token_start];

		int tokens_count = controller[doc].doc_token_count;
		int step_count = tokens_count/numBags;
		int remain = tokens_count - step_count * numBags;
		int index = tkn *  TOKEN_MAX_SIZE_PLUS_END;
		if (tkn < remain )
			step_count += 1;

		int *str;
		int step_size = numBags * TOKEN_MAX_SIZE_PLUS_END;

		int *token; 
		token = (int *)&tokens[TOKEN_MAX_SIZE_PLUS_END * tkn];
		int ratio = sizeof(int)/sizeof(char);
		for(int i=0; i< step_count; i++, index+=step_size)
		{
			int tokenLength = token_length_base[index/TOKEN_MAX_SIZE_PLUS_END]/ratio + 1;
			str = (int *)&base[index];
			// copy to shared memory first
			for (int j = 0; j != tokenLength; j++)
				token[j] = str[j];
			bool changed = ToLowerCase( (char *)token);
			changed |= Clean( (char *)token);
			changed |= stripPrefixes((char *)token);
			changed |= stripSuffixes((char *)token);
			if (changed){
				token_length_base[index/TOKEN_MAX_SIZE_PLUS_END] = strLen((char *)token);
				strCpy(&base[index], (char *)token);
			}
		}
	}
	return;
}

void MakeDocHashWrapper( char *host_local, unsigned int *token_length_host, CalcFreqController *token_division_controller_host, MyHashMapElement **hash_doc_token_sub_tables, MyHashMapElement **hash_doc_token_tables, 
		int sub_table_size, int table_size, int docs_count, int g, int b, int maxRows, int i, int tokens_count)
{
	if(i==0)
	{
		//hipMalloc	
		hipMallocPitch(&hash_doc_token_sub_tables_device, &pitch1, sub_table_size * sizeof(MyHashMapElement), MAX_GRID_SIZE);
		hipMallocPitch(&hash_doc_token_tables_device, &pitch2, table_size * sizeof(MyHashMapElement), docs_count);
	}

	dbg printf("loop %d, pitch2 %d\n", i, pitch2);

	//kernel call
	MakeDocHash<<<g, b>>>(device_local, token_length_device, &(token_division_controller_device[i]), hash_doc_token_sub_tables_device, (MyHashMapElement *)((char *)hash_doc_token_tables_device + i*pitch2), sub_table_size, table_size, maxRows, pitch1, pitch2);
	MakeDocHash2<<<g, b>>>(device_local, token_length_device, &(token_division_controller_device[i]), hash_doc_token_sub_tables_device, (MyHashMapElement *)((char *)hash_doc_token_tables_device + i*pitch2), sub_table_size, table_size, maxRows, pitch1, pitch2);

	if(maxRows != 16)
	{
		//cuda Memcpy
		hipMemcpy(host_local, device_local, tokens_count * sizeof(char) * TOKEN_MAX_SIZE_PLUS_END, hipMemcpyDeviceToHost);
		hipMemcpy(token_length_host, token_length_device, tokens_count * sizeof(unsigned int), hipMemcpyDeviceToHost);		
		
	}
}

__global__ void MakeDocHash(char *dev_mem, unsigned int *token_length, CalcFreqController *controller, 
		MyHashMapElement *hash_doc_token_sub_tables, MyHashMapElement *hash_doc_token_tables, int sub_table_size, int table_size, int maxRows, size_t pitch1, size_t pitch2)
{
	int maxCols = HASH_DOC_TOKEN_NUM_THREADS;
	int col = threadIdx.x;//z
	int row = blockIdx.x;//x
	if(col < HASH_DOC_TOKEN_NUM_THREADS && row < maxRows)
	{
		char *token_base = &dev_mem[controller[row].doc_token_start * TOKEN_MAX_SIZE_PLUS_END];
		unsigned int *token_length_base = &token_length[controller[row].doc_token_start];
		MyHashMapElement *hash_doc_token_sub_table;
		hash_doc_token_sub_table = (MyHashMapElement *)((char*)hash_doc_token_sub_tables + row * pitch1) + (sub_table_size * col / HASH_DOC_TOKEN_NUM_THREADS);
		MyHashMapElement *hash_doc_token_table;
		hash_doc_token_table = (MyHashMapElement *)((char *) hash_doc_token_tables + row * pitch2);

		{// clear the doc hash sub table in each thread
			initHashTableCuda(hash_doc_token_sub_table, HASH_DOC_TOKEN_SUB_TABLE_SIZE, HASH_DOC_TOKEN_BUCKET_SUB_SIZE);
			// clear the doc hash table
			int bucketsPerThread = HASH_DOC_TOKEN_TABLE_SIZE / maxCols;//256/64 = 4
			if (col < HASH_DOC_TOKEN_TABLE_SIZE % maxCols)
				bucketsPerThread += 1;

			MyHashMapElement *bucket = (MyHashMapElement *)hash_doc_token_table + col * HASH_DOC_TOKEN_BUCKET_SIZE;
			for (int i = 0; i != bucketsPerThread; i++)
			{
				bucket->countInBuc = 0;
				dbg{
					bucket->key = 0xDEADBEEF;
					bucket->subkey = 0;
					bucket->freq = 0;
					bucket->tokenLength = 0;
					for (int j = 1; j != HASH_DOC_TOKEN_BUCKET_SIZE; j++)
					{
						(bucket+j)->countInBuc = 0;
						(bucket+j)->freq = j;
						(bucket+j)->subkey = 0;
						(bucket+j)->key = 0xDEADBEAF;
						(bucket+j)->tokenLength = 0;
					}
				}
				bucket += maxCols * HASH_DOC_TOKEN_BUCKET_SIZE;
			}
		}

		int tokens_count = controller[row].doc_token_count;
		int step_count = tokens_count/maxCols;
		int remain = tokens_count - step_count * maxCols;
		int index = col *  TOKEN_MAX_SIZE_PLUS_END;
		if (col < remain )
			step_count += 1;

		//    int *str;
		int step_size = maxCols * TOKEN_MAX_SIZE_PLUS_END;

		for(int i=0; i< step_count; i++, index+=step_size)
		{
			unsigned long key  = computeHashCuda(&token_base[index]);
			insertElementCuda(hash_doc_token_sub_table, key, HASH_DOC_TOKEN_SUB_TABLE_SIZE_LOG2, HASH_DOC_TOKEN_BUCKET_SUB_SIZE, token_length_base[index/TOKEN_MAX_SIZE_PLUS_END], 1);
		}
		//	dbg printf("Done %d,%d\n",row,col);
	}
	return;
}

__global__ void MakeDocHash2(char *dev_mem, unsigned int *token_length, CalcFreqController *controller, 
		MyHashMapElement *hash_doc_token_sub_tables, MyHashMapElement *hash_doc_token_tables, int sub_table_size, int table_size, int maxRows, size_t pitch1, size_t pitch2)
{
	int col = threadIdx.x;//z
	int row = blockIdx.x;//x
	if(col < HASH_DOC_TOKEN_NUM_THREADS && row < maxRows)
	{
		MyHashMapElement *hash_doc_token_sub_table;
		hash_doc_token_sub_table = (MyHashMapElement *)((char*) hash_doc_token_sub_tables + row * pitch1);
		__shared__ MyHashMapElement *hash_doc_token_table;
		hash_doc_token_table = (MyHashMapElement *)((char*) hash_doc_token_tables + row * pitch2);
		hash_doc_token_sub_table += (sub_table_size * col / HASH_DOC_TOKEN_NUM_THREADS);

		// merge sub tables into one doc hash table
		hash_doc_token_sub_table = (MyHashMapElement *)((char*) hash_doc_token_sub_tables + row * pitch1);
		hash_doc_token_sub_table += (col * HASH_DOC_TOKEN_BUCKET_SUB_SIZE);
		for (int i = 0; i != HASH_DOC_TOKEN_NUM_THREADS; i++)
		{
			MyHashMapElement *bucket = hash_doc_token_sub_table;
			int numInBucket = bucket->countInBuc;
			while(numInBucket--)
			{
				unsigned long key = bucket->key;
				insertElementCuda(hash_doc_token_table, key, HASH_DOC_TOKEN_TABLE_SIZE_LOG2, HASH_DOC_TOKEN_BUCKET_SIZE, bucket->tokenLength, bucket->freq);
				bucket++;
			}
			hash_doc_token_sub_table += HASH_DOC_TOKEN_SUB_TABLE_SIZE * HASH_DOC_TOKEN_BUCKET_SUB_SIZE;
		}
	}
	return;
}

__global__ void InitOccTable(MyHashMapElement *occ_hash_table)
{
	int maxCols = 32;
	int col = threadIdx.x;//z
	int row = blockIdx.x;//x
	if(col < maxCols && row < HASH_DOC_TOKEN_TABLE_SIZE/32)
	{
		MyHashMapElement *bucket = &occ_hash_table[((row * maxCols ) + col) * OCC_HASH_TABLE_BUCKET_SIZE];
		bucket->countInBuc = 0;
		dbg{
			bucket->key = 0xDEADBEEF;
			bucket->freq = 0;
			bucket->tokenLength = 0;
			bucket->subkey = 0;
			for (int j = 1; j < OCC_HASH_TABLE_BUCKET_SIZE; j++)
			{
				bucket[j].countInBuc = 0; 
				bucket[j].key = 0xDEADBEEF;
				bucket[j].freq = 0;
				bucket[j].tokenLength = 0;
				bucket[j].subkey = 0;
			}
		}
	}
}

void AddToOccTableWrapper(MyHashMapElement **hash_doc_token_tables, MyHashMapElement *occ_hash_table, int numDocs, int occ_table_size, int g, int b, int table_size)
{
	//hipMalloc
	hipMalloc(&occ_hash_table_device, occ_table_size * sizeof(MyHashMapElement));

	//hipMemcpy
	hipMemcpy(occ_hash_table_device, occ_hash_table, occ_table_size * sizeof(MyHashMapElement), hipMemcpyHostToDevice);

	//kernel call
	InitOccTable<<<g, b>>>(occ_hash_table_device);
	AddToOccTable<<<g, b>>>(hash_doc_token_tables_device, occ_hash_table_device, numDocs, pitch2);

	//hipMemcpy	
	hipMemcpy(occ_hash_table, occ_hash_table_device, occ_table_size * sizeof(MyHashMapElement),hipMemcpyDeviceToHost);	
}

__global__ void AddToOccTable(MyHashMapElement *hash_doc_token_tables, MyHashMapElement *occ_hash_table, int numDocs, size_t pitch2)
{
	int maxCols = 32;
	int col = threadIdx.x;//z
	int row = blockIdx.x;//x
	if(col < maxCols && row < HASH_DOC_TOKEN_TABLE_SIZE/32)
	{
		for (int i = 0; i != numDocs; i++)
		{
			MyHashMapElement *hash_doc_token_table = (MyHashMapElement *)((char*)hash_doc_token_tables + i * pitch2);
			MyHashMapElement *bucket = &hash_doc_token_table[(row * maxCols + col) * HASH_DOC_TOKEN_BUCKET_SIZE];
			int numInBucket = bucket->countInBuc;
			while (numInBucket--)
			{
				unsigned long key = bucket->key;
				insertElementCuda(occ_hash_table, key, OCC_HASH_TABLE_SIZE_LOG2, OCC_HASH_TABLE_BUCKET_SIZE, bucket->tokenLength, 1);
				bucket++;
			}
		}
	}
}

__global__ void CalcTfIdf(CalcFreqController *controller,  MyHashMapElement *hash_doc_token_tables, MyHashMapElement *occ_hash_table, int docs_count, float *bucket_sqrt_sum, size_t pitch2)
{
	int maxCols = HASH_DOC_TOKEN_TABLE_SIZE;
	int col = threadIdx.x;//z
	int row = blockIdx.x;//x
	if(row < docs_count && col < maxCols)
	{
		int token_doc_count = controller[row].doc_token_count;
		// 1. calculate the un-normalized tfidf
		MyHashMapElement *bucket = (MyHashMapElement *)((char *)hash_doc_token_tables + row * pitch2);
		bucket += col * HASH_DOC_TOKEN_BUCKET_SIZE;
		int numInBucket = bucket->countInBuc;
		__shared__ float bucketSqrtSum[HASH_DOC_TOKEN_TABLE_SIZE]; 
		bucketSqrtSum[col] = 0.0f;
		while (numInBucket--)
		{
			unsigned long key = bucket->key;
			int occ = findElementCuda(occ_hash_table, key, OCC_HASH_TABLE_SIZE_LOG2, OCC_HASH_TABLE_BUCKET_SIZE, bucket->tokenLength);
			if (occ != 0)  // we should be able to find it in the occ table
			{
				float tf = (float)bucket->freq/token_doc_count;
				float idf = log(float(docs_count)/occ);
				bucket->tfidf = tf * idf;
				bucketSqrtSum[col] += bucket->tfidf * bucket->tfidf;
				dbg {
					bucket->subkey = occ;
				}
			}
			bucket++;
		}
		__syncthreads();
		if(col == 0)
		{
			float sum = 0.0f;
			for(int i = 0; i < maxCols; i++)
				sum += bucketSqrtSum[i];
			bucket_sqrt_sum[row] = sqrt(sum);
		}
	}
}

__global__ void CalcTfIdf2(CalcFreqController *controller,  MyHashMapElement *hash_doc_token_tables, MyHashMapElement *occ_hash_table, int docs_count, float *bucket_sqrt_sum, size_t pitch2)
{
	int maxCols = HASH_DOC_TOKEN_TABLE_SIZE;
	int col = threadIdx.x;//z
	int row = blockIdx.x;//x
	if(row < docs_count && col < maxCols)
	{
		// pthread_barrier_wait();
		// normalize
		float magnitude = bucket_sqrt_sum[row];
		MyHashMapElement *bucket = (MyHashMapElement *)((char *)hash_doc_token_tables + row * pitch2);
		bucket += col * HASH_DOC_TOKEN_BUCKET_SIZE;
		int numInBucket = bucket->countInBuc;
		while (numInBucket--)
		{
			float tfidf = (float)bucket->tfidf;
			tfidf = tfidf / magnitude;
			bucket->tfidf = tfidf;
			bucket++;
		}
	}
}

void CalcTfidfWrapper(CalcFreqController *token_division_controller_host, MyHashMapElement **hash_doc_token_tables_host, MyHashMapElement *occ_hash_table_remote, int docs_count, float *bucket_sqrt_sum, int g, int b, int table_size)
{
	//hipMalloc
	hipMalloc(&bucket_sqrt_sum_device, HASH_DOC_TOKEN_TABLE_SIZE * sizeof(float));

	//kernel calls
	CalcTfIdf<<<g, b>>>(token_division_controller_device, hash_doc_token_tables_device, occ_hash_table_device, docs_count, bucket_sqrt_sum_device, pitch2);
	CalcTfIdf2<<<g, b>>>(token_division_controller_device, hash_doc_token_tables_device, occ_hash_table_device, docs_count, bucket_sqrt_sum_device, pitch2);

	//hipMemcpy
	for(int j=0; j< docs_count;j++)
		hipMemcpy(hash_doc_token_tables_host[j], (MyHashMapElement *)((char*)hash_doc_token_tables_device + j * pitch2), table_size * sizeof(MyHashMapElement), hipMemcpyDeviceToHost);

}

__global__ void CalcSimilarities(MyHashMapElement *hash_doc_token_tables, MyHashMapElement *occ_hash_table_remote, float *similarity_matrix, int docs_count, size_t pitch2)
{
	int col = threadIdx.x;
	int row = blockIdx.x;
	int row2 = blockIdx.y;
	int maxCols = HASH_DOC_TOKEN_TABLE_SIZE;
	if(col < HASH_DOC_TOKEN_TABLE_SIZE && row < docs_count && row2 < docs_count)
	{
		MyHashMapElement *hashDoc_token_table1 = (MyHashMapElement *)((char *)hash_doc_token_tables + row * pitch2); 
		MyHashMapElement *hashDoc_token_table2 = (MyHashMapElement *)((char *)hash_doc_token_tables + row2 * pitch2); 
		__shared__ float sim_sum[HASH_DOC_TOKEN_TABLE_SIZE];
		sim_sum[col] = 0.0f;
		MyHashMapElement *bucket1 = hashDoc_token_table1 + col * HASH_DOC_TOKEN_BUCKET_SIZE;

		int num_ele_1 = bucket1->countInBuc;
		while (num_ele_1--)
		{
			MyHashMapElement *bucket2 = hashDoc_token_table2 + col * HASH_DOC_TOKEN_BUCKET_SIZE;
			int num_ele_2 = bucket2->countInBuc;
			int find = 0;
			while (num_ele_2--)
			{
				if ((bucket2->key == bucket1->key) && (bucket2->tokenLength == bucket1->tokenLength))
				{
					find = 1;
					break;
				}
				bucket2++;
			}
			if (find)
				sim_sum[col] += bucket1->tfidf * bucket2->tfidf;

			bucket1++;
		}
		__syncthreads();
		if(col == 0)
		{
			float sum = 0.0f;
			for(int i = 0; i < maxCols; i++)
				sum += sim_sum[i];
			similarity_matrix[docs_count * row + row2] = sum;
		}
	}
}

void CalcSimilaritiesWrapper(MyHashMapElement **hash_doc_token_tables_host, MyHashMapElement *occ_hash_table_remote, float *doc_similarity_matrix_host, int docs_count, int g, int b)
{
	//hipMalloc
	hipMalloc(&doc_similarity_matrix_device, docs_count * docs_count * sizeof(float));

	dim3 threadsPerBlock(b, b);
	dim3 numBlocks(g/2,g/2);
	//kernel calls
	CalcSimilarities<<<numBlocks, b>>>(hash_doc_token_tables_device, occ_hash_table_device, doc_similarity_matrix_device, docs_count, pitch2);

	//hipMemcpy
	hipMemcpy(doc_similarity_matrix_host, doc_similarity_matrix_device, docs_count * docs_count * sizeof(float),hipMemcpyDeviceToHost);
}

__global__ void SortSimilarities2(float *similarity_matrix, int *rank_matrix, int docs_count, float *simbase, int *rankbase)
{
	int col = threadIdx.x;
	int row = blockIdx.x;
	if(col < docs_count && row < docs_count)
	{
		simbase = (float *)similarity_matrix+row*docs_count;
		rankbase = (int *)rank_matrix+row * docs_count;
		float my_value = *((float *)simbase+col);
		int myRank = 0;
		for (int i = 0; i != docs_count; i++)
		{
			if (i == col) 
				continue;
			if (*((float *)simbase+i) > my_value)
				myRank = myRank + 1;
		}

		*((int *)rankbase+col) = myRank;
	}
}

void SortSimilarities2Wrapper(float *doc_similarity_matrix_host, int *doc_rank_matrix_host, int docs_count, int g, int b)
{
	//hipMalloc
	hipMalloc(&doc_rank_matrix_device, docs_count * docs_count * sizeof(int));
	hipMalloc(&simbase, docs_count*sizeof(float));
	hipMalloc(&rankbase, docs_count*sizeof(int));
	
	//kernel call
	SortSimilarities2<<<g, b>>>(doc_similarity_matrix_device, doc_rank_matrix_device, docs_count, simbase, rankbase);

	//hipMemcpy
	hipMemcpy(doc_rank_matrix_host, doc_rank_matrix_device, docs_count * docs_count * sizeof(int), hipMemcpyDeviceToHost);

	//hipFree
	hipFree(&doc_rank_matrix_device);
	hipFree(&doc_similarity_matrix_device);
	hipFree(&hash_doc_token_tables_device);
	hipFree(&occ_hash_table_device);
	hipFree(&token_division_controller_device);
	hipFree(&bucket_sqrt_sum_device);
	hipFree(device_local);
	hipFree(token_length_device);
	hipFree(hash_doc_token_sub_tables_device);
}


